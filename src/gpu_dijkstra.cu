#include <hip/hip_runtime.h>
#include <cstdio>
#include <algorithm>
#include <climits>

extern "C" {

__global__ void dijkstra_step(int* grid, int* dist, int* updated, int rows, int cols){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx >= rows*cols) return;
    if(updated[idx]==0) return;

    int r = idx/cols, c = idx%cols;
    updated[idx] = 0;
    int dr[] = {-1,1,0,0};
    int dc[] = {0,0,-1,1};
    for(int k=0;k<4;k++){
        int nr=r+dr[k], nc=c+dc[k];
        if(nr>=0 && nr<rows && nc>=0 && nc<cols){
            int nidx = nr*cols+nc;
            int new_dist = dist[idx] + grid[nidx];
            int old = atomicMin(&dist[nidx], new_dist);
            if(new_dist < old) updated[nidx] = 1;
        }
    }
}

bool gpu_available(){
    int n=0;
    hipGetDeviceCount(&n);
    return n>0;
}

// GPU wrapper
double gpu_dijkstra_time_ms(int* grid_h, int rows, int cols, int start, int goal, int* dist_h){
    int N = rows*cols;
    int *grid_d=nullptr,*dist_d=nullptr,*upd_d=nullptr;
    hipMalloc(&grid_d,sizeof(int)*N);
    hipMalloc(&dist_d,sizeof(int)*N);
    hipMalloc(&upd_d,sizeof(int)*N);

    hipMemcpy(grid_d,grid_h,sizeof(int)*N,hipMemcpyHostToDevice);
    std::vector<int> dist(N,INT_MAX), updated(N,0);
    dist[start]=0; updated[start]=1;
    hipMemcpy(dist_d, dist.data(), sizeof(int)*N,hipMemcpyHostToDevice);
    hipMemcpy(upd_d, updated.data(), sizeof(int)*N,hipMemcpyHostToDevice);

    int block=256, gridSize=(N+block-1)/block;
    hipEvent_t start_ev, stop_ev;
    hipEventCreate(&start_ev); hipEventCreate(&stop_ev);
    hipEventRecord(start_ev);

    for(int iter=0; iter<N; iter++){
        dijkstra_step<<<gridSize,block>>>(grid_d, dist_d, upd_d, rows, cols);
        hipDeviceSynchronize();
    }

    hipEventRecord(stop_ev);
    hipEventSynchronize(stop_ev);
    float ms=0; hipEventElapsedTime(&ms,start_ev,stop_ev);

    hipMemcpy(dist_h, dist_d, sizeof(int)*N, hipMemcpyDeviceToHost);
    hipFree(grid_d); hipFree(dist_d); hipFree(upd_d);
    hipEventDestroy(start_ev); hipEventDestroy(stop_ev);
    return double(ms);
}

} // extern "C"
